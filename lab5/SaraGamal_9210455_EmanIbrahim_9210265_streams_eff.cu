#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h> 


#define block_dim 256
#define streams_num 8


// Kernel: Efficient prefix sum 
__global__ void efficient_prefix_sum(int *in, int *out, int *block_finished, int n) {
    __shared__ int sh_in[2*256];
    int i = threadIdx.x;
    int start = 2 * blockIdx.x * blockDim.x;

    // load 1st element
    if (start + i < n)
        sh_in[i] = in[start + i];
    else
        sh_in[i] = 0;

    // load 2nd element
    if (start+ i +blockDim.x  < n)
        sh_in[i+blockDim.x] = in[start + i+ blockDim.x ];
    else
        sh_in[blockDim.x + i] = 0;

    // reduction
    for (int stride = 1; stride <= blockDim.x; stride *= 2) {
        __syncthreads();
        int indx = (i + 1) * 2 * stride - 1;
        if (indx < 2 * blockDim.x) {
            sh_in[indx] += sh_in[indx - stride];
        }
    }

    if (i == 0) {
        sh_in[2 * blockDim.x - 1] = 0;
    }

    for (int stride = blockDim.x; stride >= 1; stride /= 2) {
        __syncthreads();
        int indx = (i + 1) * 2 * stride - 1;
        if (indx < 2 * blockDim.x) {
            int temp = sh_in[indx - stride];
            sh_in[indx - stride] = sh_in[indx];
            sh_in[indx] += temp;
        }
    }

    __syncthreads();
    if (start + i < n) {
        out[start + i] = sh_in[i] + in[start + i]; 
        __threadfence();
    }
    if (start+ i +blockDim.x  < n) {
        out[start+ i +blockDim.x ] = sh_in[blockDim.x + i] + in[start + i +blockDim.x];
        __threadfence();
    }

    while (blockIdx.x > 0 && atomicAdd(&block_finished[blockIdx.x-1],0) == 0) {}
    __syncthreads();

    if (start + i < n && blockIdx.x > 0) {
        out[start + i] += out[2 * blockIdx.x * blockDim.x - 1]; 
    }
    if (start +i+ blockDim.x  < n && blockIdx.x > 0) {
        out[start +i+ blockDim.x] += out[2 * blockIdx.x * blockDim.x - 1]; 
    }

    if (threadIdx.x == 0) {
        __threadfence();
        atomicAdd(&block_finished[blockIdx.x],1);
    }
}

// Kernel: Add previous streams' sums
__global__ void add_stream_sums(int *d_out, int stream_sum, int n, int offset, int size) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < size && (i + offset) < n) {
        d_out[i + offset] += stream_sum;
    }
}

// Host function to perform prefix sum with streams
void prefix_sum_streams(int *h_in, int *h_out, int n) {
    int *d_in, *d_out;
    int **d_block_finished; // Per-stream block_finished arrays
    int *h_block_sums;      // Host storage for last prefix sum per stream
    hipStream_t streams[streams_num];
    hipEvent_t events[streams_num];
    int chunk_size = (n + streams_num - 1) / streams_num;

    // Allocate device memory
    hipMalloc(&d_in, n * sizeof(int));
    hipMalloc(&d_out, n * sizeof(int));
    h_block_sums = (int *)malloc(streams_num * sizeof(int));
    d_block_finished = (int **)malloc(streams_num * sizeof(int *));

    // Allocate per-stream block_finished arrays
    int *h_block_finished_ptrs[streams_num];
    for (int i = 0; i < streams_num; i++) {
        int size = min(chunk_size, n - i * chunk_size);
        int local_blocks = (size + 2 * block_dim - 1) / (2 * block_dim);
        hipMalloc(&h_block_finished_ptrs[i], local_blocks * sizeof(int));
        hipMemset(h_block_finished_ptrs[i], 0, local_blocks * sizeof(int));
    }

    // Create streams and events
    for (int i = 0; i < streams_num; i++) {
        hipStreamCreate(&streams[i]);
        hipEventCreate(&events[i]);
    }

    // Process each stream
    for (int i = 0; i < streams_num; i++) {
        int offset = i * chunk_size;
        int size = min(chunk_size, n - offset);
        if (size > 0) {
            int local_blocks = (size + 2 * block_dim - 1) / (2 * block_dim);

            // Copy input data
            hipMemcpyAsync(d_in + offset, h_in + offset, size * sizeof(int), hipMemcpyHostToDevice, streams[i]);

            // Launch efficient prefix sum kernel
            efficient_prefix_sum<<<local_blocks, block_dim, 2 * block_dim * sizeof(int), streams[i]>>>(d_in + offset, d_out + offset, h_block_finished_ptrs[i], size);
            hipGetLastError();

            // Copy last prefix sum to host
            if (local_blocks > 0) {
                hipMemcpyAsync(&h_block_sums[i], d_out + offset + size - 1, sizeof(int), hipMemcpyDeviceToHost, streams[i]);
            }

            hipEventRecord(events[i], streams[i]);
        }
    }

    // Add previous streams' sums
    for (int i = 1; i < streams_num; i++) {
        int offset = i * chunk_size;
        int size = min(chunk_size, n - offset);
        if (size > 0) {
            int local_blocks = (size + 2 * block_dim - 1) / (2 * block_dim);

            // Wait for previous stream
            hipStreamWaitEvent(streams[i], events[i - 1], 0);

            // Compute cumulative sum of previous streams
            int cumulative_sum = 0;
            for (int j = 0; j < i; j++) {
                hipStreamSynchronize(streams[j]);
                cumulative_sum += h_block_sums[j];
            }

            // Launch kernel to add cumulative sum
            add_stream_sums<<<local_blocks, block_dim * 2, 0, streams[i]>>>(d_out, cumulative_sum, n, offset, size);
            hipGetLastError();
        }
    }

    // Copy output to host
    for (int i = 0; i < streams_num; i++) {
        int offset = i * chunk_size;
        int size = min(chunk_size, n - offset);
        if (size > 0) {
            hipMemcpyAsync(h_out + offset, d_out + offset, size * sizeof(int), hipMemcpyDeviceToHost, streams[i]);
        }
    }

    // Synchronize and clean up
    hipDeviceSynchronize();
    for (int i = 0; i < streams_num; i++) {
        hipStreamDestroy(streams[i]);
        hipEventDestroy(events[i]);
        hipFree(h_block_finished_ptrs[i]);
    }
    hipFree(d_in);
    hipFree(d_out);
    free(h_block_sums);
    free(d_block_finished);
}

void check(int *h_in, int *h_out, int n) {
    int sum = 0;
    for (int i = 0; i < n; i++) {
        sum += h_in[i];
        if (h_out[i] != sum) {
            printf("failed\n");
            return;
        }
    }
    printf("passed\n");
}

int main() {
    int n;
    int *h_in, *h_out;

    FILE *file = fopen("input2.txt", "r");
    if (!file) {
        perror("Can't open the input file");
        return -1;
    }
    fscanf(file, "%d", &n);
    h_in = (int *)malloc(n * sizeof(int));
    for (int i = 0; i < n; i++) {
        fscanf(file, "%d", &h_in[i]);
    }
    fclose(file);

    h_out = (int *)malloc(n * sizeof(int));

    // Timing starts here
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);

    prefix_sum_streams(h_in, h_out, n);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Total execution time including all streams: %.6f ms\n", milliseconds);

    // Clean up the timing events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    check(h_in, h_out, n);

    FILE *out_file = fopen("output.txt", "w");
    if (!out_file) {
        perror("Can't open the output file");
        return -1;
    }
    for (int i = 0; i < n; i++) {
        fprintf(out_file, "%d ", h_out[i]);
    }
    fclose(out_file);

    free(h_in);
    free(h_out);
    return 0;
}