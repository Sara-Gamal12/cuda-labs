#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <fstream>
#include <assert.h>
#include <string.h>
#include <dirent.h>
#include <vector>
#include <string>
#include <float.h>
#include<iostream>
#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

#include <sys/stat.h>
__constant__ float c_mask[256];


void save_images(const char* output_folder, float* output_data,   int width, int height, int channels,  int batch_size,std::vector<std::string> input_paths,int batch_start) {
    // Create output directory if it doesn't exist
    mkdir(output_folder, 0777);


    // Process each image in the current batch
    for (int i = 0; i < batch_size; i++) {

        // Extract filename from input path
        std::string path = input_paths[ i+batch_start];

        size_t last_slash = path.find_last_of("/\\");
        std::string filename = (last_slash == std::string::npos) ? path : path.substr(last_slash + 1);

        // Create output path (preserve extension)
        std::string output_path = std::string(output_folder) + "/conv_" + filename;

        // Allocate memory for output image (convert from float to uint8)
        unsigned char* image_data = (unsigned char*)malloc(width * height*channels );

      float min_pixel = FLT_MAX;
       float max_pixel = -FLT_MAX;

  for (int k=0;k<channels;k++)
       for (int y = 0; y < height; y++) {
            for (int x = 0; x < width; x++) {

          int output_idx = ((i*channels+k )* height * width ) +        (y * width ) +   (x ) ;
            if (output_data[output_idx] < min_pixel)
                min_pixel = output_data[output_idx];
            if (output_data[output_idx] > max_pixel)
                max_pixel = output_data[output_idx];
        }}
        // Convert and normalize output data
        for(int k=0;k<channels;k++)
        for (int y = 0; y < height; y++) {
            for (int x = 0; x < width; x++) {
              {
                    // Calculate indices (NHWC layout)
                    int output_idx = ((i*channels+k ) * height * width ) +
                                   (y * width ) +
                                   (x ) ;

                    float pixel_val = output_data[output_idx];

                    pixel_val=static_cast<unsigned char>(255.0f *(pixel_val-min_pixel)/(max_pixel-min_pixel));
                   image_data[((y * width + x)*channels+k) ] = pixel_val;
                }
            }
        }

        // Save image (preserve original format)
        std::string ext = filename.substr(filename.find_last_of(".") + 1);
        int success = 0;
        if (ext == "png") {
            success = stbi_write_png(output_path.c_str(), width, height, channels, image_data, width * channels);
        }
        else if (ext == "jpg" || ext == "jpeg") {
            success = stbi_write_jpg(output_path.c_str(), width, height, channels, image_data, 90);  // 90% quality
        }
        else {
            printf("Unsupported output format for %s, defaulting to PNG\n", output_path.c_str());
            success = stbi_write_png(output_path.c_str(), width, height, 1, image_data, width * 1);
        }

        if (!success) {
            printf("Failed to save image %s\n", output_path.c_str());
        }

        free(image_data);
    }
}

__global__ void conv3D_basic(const uint8_t *input, int width, int height, int depth,int batch_size, float *output, float *mask, int maskWidth)
 {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int batch_index= threadIdx.z+blockIdx.z * blockDim.z;


    if (col >= width || row >= height||batch_index>=batch_size ) return;


   for (int channel=0;channel<depth;channel++)
{
   float sum = 0.0f;
    for (int i = 0; i < maskWidth; ++i) {
        for (int j =0; j < maskWidth; ++j) {

            int curr_row = row+i-maskWidth/2;
            int curr_col = col+j-maskWidth/2;
            if(curr_col<width&& curr_row<height&&curr_col>=0&&curr_row>=0)
            {

             {
               sum+=c_mask[i*maskWidth+j]*static_cast<float>(input[batch_index*height*width*depth  +  curr_row*width*depth+curr_col*depth+channel]);

}
            }
        }
    }
      int outIdx = (batch_index*depth+channel)*height*width+row*width+col;
               output[outIdx] = sum;}

}


float* read_mask(const char* file_path, int& maskWidth) {
    FILE* file = fopen(file_path, "r");
    if (!file) {
        fprintf(stderr, "Error: Could not open mask file %s\n", file_path);
        return nullptr;
    }

    // Read mask dimension (first line)
    if (fscanf(file, "%d", &maskWidth) != 1) {
        fprintf(stderr, "Error: Could not read mask dimension from %s\n", file_path);
        fclose(file);
        return nullptr;
    }

    float* mask = (float*)malloc(maskWidth * maskWidth * sizeof(float));
    if (!mask) {
        fprintf(stderr, "Error: Memory allocation failed for mask\n");
        fclose(file);
        return nullptr;
    }

    // Read mask values (subsequent lines)
    for (int i = 0; i < maskWidth; i++) {
        for (int j = 0; j < maskWidth; j++) {
            if (fscanf(file, "%f", &mask[i * maskWidth + j]) != 1) {
                fprintf(stderr, "Error: Invalid mask data at row %d, column %d\n", i+1, j+1);
                free(mask);
                fclose(file);
                return nullptr;
            }
        }
    }

    fclose(file);
    return mask;
}


uint8_t* load_images(const char* folder_path, int& width, int& height, int& channels, int batch_size,int & num_images, std::vector<std::string>& image_paths) {
    DIR *dir;
    struct dirent *ent;

    if ((dir = opendir(folder_path)) != NULL) {
        while ((ent = readdir(dir)) != NULL) {
            std::string filename = ent->d_name;
            if (filename.find(".jpg") != std::string::npos ||
                filename.find(".jpeg") != std::string::npos ||
                filename.find(".png") != std::string::npos) {
                image_paths.push_back(std::string(folder_path) + "/" + filename);
            }
        }
        closedir(dir);
    } else {
        perror("Could not open directory");
        return nullptr;
    }

    if (image_paths.empty()) {
        printf("No images found in %s\n", folder_path);
        return nullptr;
    }

   num_images=image_paths.size();
   uint8_t* h_input;
    // Load images into batch
    for (int i = 0; i < image_paths.size(); i++) {
        int img_width, img_height, img_channels;
        unsigned char* image_data = stbi_load(image_paths[i].c_str(), &img_width, &img_height, &img_channels, 0);

        if(i==0)
        {
           height=img_height;
        width=img_width;
        channels=img_channels;
           size_t input_size = image_paths.size() * height * width * channels * sizeof(uint8_t);
            h_input = (uint8_t*)malloc(input_size);

        }
        if (!image_data) {
            printf("Failed to load image: %s\n", image_paths[i].c_str());
            continue;
        }


        // Copy image data to batch (NHWC layout)
        for (int y = 0; y < height; y++) {
            for (int x = 0; x < width; x++) {
                for (int c = 0; c < channels; c++) {
                    int src_idx = (y * width + x) * channels + c;
                    int dst_idx = (i * height * width * channels) +
                                 (y * width * channels) +
                                 (x * channels) + c;
                    h_input[dst_idx] = image_data[src_idx];
                }
            }
        }

        stbi_image_free(image_data);
    }

    return h_input;
}

int main(int argc, char** argv)
{

   if (argc != 5) {
        printf("arguments are incorrect");
        return 1;
    }
    const char* input_folder = argv[1];
    const char* output_folder = argv[2];
    int batch_size = atoi(argv[3]);
    const char* mask_file = argv[4];



    int maskWidth;
    float*h_mask=read_mask(mask_file,maskWidth);
    if(!h_mask)
    {
      return 1;

    }


    int height,width,depth;
    uint8_t* h_input;
    int num_images;
    std::vector<std::string> input_paths;
    h_input=load_images(input_folder,width,height,depth,batch_size, num_images,input_paths);






    uint8_t* d_input;
    float* d_output;
    float* d_mask;

    //Allocate


    hipMalloc(&d_mask, maskWidth * maskWidth * sizeof(float));
    hipMemcpy(d_mask, h_mask, maskWidth * maskWidth * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(c_mask), h_mask, maskWidth * maskWidth * sizeof(float));

    for(int batch_start=0;batch_start<num_images;batch_start+=batch_size)
    {

        int current_batch_size = (batch_start + batch_size > num_images) ? num_images - batch_start : batch_size;
        size_t input_size = current_batch_size * height * width * sizeof(uint8_t)*depth;
        size_t output_size = current_batch_size * height * width * sizeof(float)*depth;;
         float* h_output = (float*)malloc(output_size);

            //copy to gpu
            hipMalloc(&d_input, input_size);
            hipMalloc(&d_output, output_size);


            hipMemcpy(d_input,  &h_input[batch_start * width * height * depth], input_size, hipMemcpyHostToDevice);

   dim3 block_size(16, 16, 1);
   dim3 grid_size(
       (width + block_size.x - 1) / block_size.x,
       (height + block_size.y - 1) / block_size.y,
       current_batch_size

   );


            conv3D_basic<<<grid_size, block_size>>>(d_input, width, height, depth, current_batch_size,
                d_output, d_mask, maskWidth);

            hipError_t err = hipGetLastError();
            if (err != hipSuccess) {
            printf("CUDA error: %s\n", hipGetErrorString(err));
            }

            hipDeviceSynchronize();  // Required to flush printf output


            hipMemcpy(h_output, d_output, output_size, hipMemcpyDeviceToHost);


            save_images(output_folder,h_output,width,height,depth,current_batch_size,input_paths,batch_start);
            hipFree(d_input);
            hipFree(d_output);
            free(h_output);


    }



  free(h_mask);
    free(h_input);
    hipFree(d_mask);

    return 0;


}
